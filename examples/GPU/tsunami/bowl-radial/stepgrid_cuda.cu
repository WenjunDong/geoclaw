#include "real.H"
#include "params.H"
#include "shallow_topo_riemann_solver.H"
#include "fused_solvers_headers.H"
#include "advance.H"
#include <hip/hip_runtime.h>

extern "C" void call_C_limited_riemann_update(
        const int cellsX, const int cellsY, const int ghostCells,
        const real startX, const real endX, const real startY, const real endY,
        const real dt,
        real* q, real* qNew, 
        real* coefficients,
        real* waveSpeedsX, real* waveSpeedsY,
        const int numStates, const int numCoefficients,
        real* cfls, const int ngrids, const int mcapa,
        const int id, const int dev_id) {

    // actually qNew holds the input old solution as well as new output solution
    // q is just temporary storage for intermediate solution between x-sweep and y-sweep

    hipStream_t stream;

    get_cuda_stream(id, dev_id, &stream);

    pdeParam param(cellsX, cellsY, ghostCells, 
            numStates, NWAVES, numCoefficients,
            startX, endX, startY, endY, dt,
            q, qNew, 
            coefficients, 
            waveSpeedsX, waveSpeedsY,
            cfls, mcapa, id, dev_id);

    param.setOrderOfAccuracy(2);

    shallow_topo_horizontal swe_h;
    shallow_topo_vertical swe_v;
    
    limiter_MC phi;
    // limiter_VanLeer phi;

    limited_Riemann_Update(param, 
            swe_h, swe_v, 
            phi,stream);

}

